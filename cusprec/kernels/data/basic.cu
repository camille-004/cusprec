
#include <hip/hip_runtime.h>
__global__ void initialize(
    float *x,
    float *A,
    float *b,
    float *rand_vals,
    int *rand_indices,
    int m,
    int n,
    int k
) {
    /*
    Generate the values for sparse signal recovery.

    Args:
        x (float*): Pointer to the sparse signal vector.
        A (float*): Pointer to the measurement matrix.
        b (float*): Pointer to the observed signal vector.
        rand_vals (float*): Pointer to the random values array.
        rand_indices (int*): Pointer to the random indices array.
        m (int): Number of rows in the measurement matrix.
        n (int): Number of columns in the measurement matrix.
        k (int): Sparsity level of the sparse signal.
    */
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < k) {
        int i = rand_indices[idx];
        x[i] = rand_vals[i * (m + 1)];
    }

    if (idx < n) {
        for (int i = 0; i < m; i++) {
            A[i * n + idx] = rand_vals[idx + i + 1];
        }

        float sum_squares = 0.0f;
        for (int i = 0; i < m; i++) {
            sum_squares += A[i * n + idx] * A[i * n + idx];
        }
        float norm = sqrtf(sum_squares);

        for (int i = 0; i < m; i++) {
            A[i * n + idx] /= norm;
        }
    }

    if (idx < m) {
        float sum = 0.0f;
        for (int j = 0; j < n; j++) {
            sum += A[idx * n + j] * x[j];
        }
        b[idx] = sum;
    }
}
